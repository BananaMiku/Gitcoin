#include "hip/hip_runtime.h"
#include <iostream>

/*
Based on
SHA-1 in C
By Steve Reid <steve@edmweb.com>
100% Public Domain

Test Vectors (from FIPS PUB 180-1)
"abc"
  A9993E36 4706816A BA3E2571 7850C26C 9CD0D89D
"abcdbcdecdefdefgefghfghighijhijkijkljklmklmnlmnomnopnopq"
  84983E44 1C3BD26E BAAE4AA1 F95129E5 E54670F1
A million repetitions of "a"
  34AA973C D4C4DAA4 F61EEB2B DBAD2731 6534016F
*/

#include <string.h>

#include <cstdint>

#include "sha1.h"


#define rol(value, bits) (((value) << (bits)) | ((value) >> (32 - (bits))))

/* blk0() and blk() perform the initial expand. */
/* I got the idea of expanding during the round function from SSLeay */
#define blk0(i) (block->l[i] = (rol(block->l[i],24)&0xFF00FF00) \
    |(rol(block->l[i],8)&0x00FF00FF))

#define blk(i) (block->l[i&15] = rol(block->l[(i+13)&15]^block->l[(i+8)&15] \
    ^block->l[(i+2)&15]^block->l[i&15],1))

/* (R0+R1), R2, R3, R4 are the different operations used in SHA1 */
#define R0(v,w,x,y,z,i) z+=((w&(x^y))^y)+blk0(i)+0x5A827999+rol(v,5);w=rol(w,30);
#define R1(v,w,x,y,z,i) z+=((w&(x^y))^y)+blk(i)+0x5A827999+rol(v,5);w=rol(w,30);
#define R2(v,w,x,y,z,i) z+=(w^x^y)+blk(i)+0x6ED9EBA1+rol(v,5);w=rol(w,30);
#define R3(v,w,x,y,z,i) z+=(((w|x)&y)|(w&x))+blk(i)+0x8F1BBCDC+rol(v,5);w=rol(w,30);
#define R4(v,w,x,y,z,i) z+=(w^x^y)+blk(i)+0xCA62C1D6+rol(v,5);w=rol(w,30);


/* Hash a single 512-bit block. This is the core of the algorithm. */

class SHA1_CTX {
public:
    uint32_t state[5];
    uint32_t count[2];
    unsigned char buffer[64];

    __device__ __host__
    SHA1_CTX() {
        state[0] = 0x67452301;
        state[1] = 0xEFCDAB89;
        state[2] = 0x98BADCFE;
        state[3] = 0x10325476;
        state[4] = 0xC3D2E1F0;
        count[0] = count[1] = 0;
    }

    __device__ __host__
    void update(const unsigned char *data, uint32_t len) {
        uint32_t i;

        uint32_t j;

        j = count[0];
        if ((count[0] += len << 3) < j)
            count[1]++;
        count[1] += (len >> 29);
        j = (j >> 3) & 63;
        if ((j + len) > 63)
        {
            memcpy(&buffer[j], data, (i = 64 - j));
            SHA1Transform(state, buffer);
            for (; i + 63 < len; i += 64)
            {
                SHA1Transform(state, &data[i]);
            }
            j = 0;
        }
        else
            i = 0;
        memcpy(&buffer[j], &data[i], len - i);
    }

    __device__ __host__
    void final(unsigned char digest[20]) {
        unsigned i;

        unsigned char finalcount[8];

        unsigned char c;

        for (i = 0; i < 8; i++)
        {
            finalcount[i] = (unsigned char) ((count[(i >= 4 ? 0 : 1)] >> ((3 - (i & 3)) * 8)) & 255);      /* Endian independent */
        }
        c = 0200;
        update(&c, 1);
        while ((count[0] & 504) != 448)
        {
            c = 0000;
            update(&c, 1);
        }
        update(finalcount, 8); /* Should cause a SHA1Transform() */
        for (i = 0; i < 20; i++)
        {
            digest[i] = (unsigned char)
                ((state[i >> 2] >> ((3 - (i & 3)) * 8)) & 255);
        }
    }
};

struct Nonce {
    unsigned char data[16];
};

__device__ __host__
void SHA1Transform(
    uint32_t state[5],
    const unsigned char buffer[64]
)
{
    uint32_t a, b, c, d, e;

    typedef union
    {
        unsigned char c[64];
        uint32_t l[16];
    } CHAR64LONG16;

    CHAR64LONG16 block[1];      /* use array to appear as a pointer */

    memcpy(block, buffer, 64);

    /* Copy context->state[] to working vars */
    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];
    /* 4 rounds of 20 operations each. Loop unrolled. */
    R0(a, b, c, d, e, 0);
    R0(e, a, b, c, d, 1);
    R0(d, e, a, b, c, 2);
    R0(c, d, e, a, b, 3);
    R0(b, c, d, e, a, 4);
    R0(a, b, c, d, e, 5);
    R0(e, a, b, c, d, 6);
    R0(d, e, a, b, c, 7);
    R0(c, d, e, a, b, 8);
    R0(b, c, d, e, a, 9);
    R0(a, b, c, d, e, 10);
    R0(e, a, b, c, d, 11);
    R0(d, e, a, b, c, 12);
    R0(c, d, e, a, b, 13);
    R0(b, c, d, e, a, 14);
    R0(a, b, c, d, e, 15);
    R1(e, a, b, c, d, 16);
    R1(d, e, a, b, c, 17);
    R1(c, d, e, a, b, 18);
    R1(b, c, d, e, a, 19);
    R2(a, b, c, d, e, 20);
    R2(e, a, b, c, d, 21);
    R2(d, e, a, b, c, 22);
    R2(c, d, e, a, b, 23);
    R2(b, c, d, e, a, 24);
    R2(a, b, c, d, e, 25);
    R2(e, a, b, c, d, 26);
    R2(d, e, a, b, c, 27);
    R2(c, d, e, a, b, 28);
    R2(b, c, d, e, a, 29);
    R2(a, b, c, d, e, 30);
    R2(e, a, b, c, d, 31);
    R2(d, e, a, b, c, 32);
    R2(c, d, e, a, b, 33);
    R2(b, c, d, e, a, 34);
    R2(a, b, c, d, e, 35);
    R2(e, a, b, c, d, 36);
    R2(d, e, a, b, c, 37);
    R2(c, d, e, a, b, 38);
    R2(b, c, d, e, a, 39);
    R3(a, b, c, d, e, 40);
    R3(e, a, b, c, d, 41);
    R3(d, e, a, b, c, 42);
    R3(c, d, e, a, b, 43);
    R3(b, c, d, e, a, 44);
    R3(a, b, c, d, e, 45);
    R3(e, a, b, c, d, 46);
    R3(d, e, a, b, c, 47);
    R3(c, d, e, a, b, 48);
    R3(b, c, d, e, a, 49);
    R3(a, b, c, d, e, 50);
    R3(e, a, b, c, d, 51);
    R3(d, e, a, b, c, 52);
    R3(c, d, e, a, b, 53);
    R3(b, c, d, e, a, 54);
    R3(a, b, c, d, e, 55);
    R3(e, a, b, c, d, 56);
    R3(d, e, a, b, c, 57);
    R3(c, d, e, a, b, 58);
    R3(b, c, d, e, a, 59);
    R4(a, b, c, d, e, 60);
    R4(e, a, b, c, d, 61);
    R4(d, e, a, b, c, 62);
    R4(c, d, e, a, b, 63);
    R4(b, c, d, e, a, 64);
    R4(a, b, c, d, e, 65);
    R4(e, a, b, c, d, 66);
    R4(d, e, a, b, c, 67);
    R4(c, d, e, a, b, 68);
    R4(b, c, d, e, a, 69);
    R4(a, b, c, d, e, 70);
    R4(e, a, b, c, d, 71);
    R4(d, e, a, b, c, 72);
    R4(c, d, e, a, b, 73);
    R4(b, c, d, e, a, 74);
    R4(a, b, c, d, e, 75);
    R4(e, a, b, c, d, 76);
    R4(d, e, a, b, c, 77);
    R4(c, d, e, a, b, 78);
    R4(b, c, d, e, a, 79);
    /* Add the working vars back into context.state[] */
    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    /* Wipe variables */
    a = b = c = d = e = 0;
    memset(block, '\0', sizeof(block));
}



/* Add padding and return the message digest. */

__device__ __host__
void SHA1(
    char *hash_out,
    const char *str,
    uint32_t len)
{
    SHA1_CTX ctx;
    unsigned int ii;

    for (ii=0; ii<len; ii+=1)
        ctx.update((const unsigned char*)str + ii, 1);
    ctx.final((unsigned char *)hash_out);
}

__device__ __host__
bool mine_single_fast(SHA1_CTX ctx, Nonce nonce) {
    char nonce_string[20];
    char hash[20];
    for(int i = 0; i < 16; i++) {
        nonce_string[30 - i * 2] = hex_chars[(nonce.data[i] & 0xf0) >> 4];
        nonce_string[31 - i * 2] = hex_chars[nonce.data[i] & 0x0f];
    }

    for(int i = 0; i < 33; i++) {
        ctx.update((const unsigned char*)nonce_string + i, 1);
    }
    ctx.final((unsigned char *)hash);

    return hash[0] || hash[1];
}

constexpr size_t THREADS_PER_BLOCK = 32;

__global__ 
void mine(SHA1_CTX original, Nonce *target, size_t limit) {
    __shared__ SHA1_CTX workspace[THREADS_PER_BLOCK];

    SHA1_CTX &ctx = workspace[threadIdx.x];

    Nonce &target = target[blockIdx.x];

    uint64_t initial_hash_value = 0;
    uint64_t hash_value = initial_hash_value;

    Nonce nonce;
    do {
        hash_value++;
        *((uint64_t*)&nonce) = hash_value;
        *((uint64_t*)&nonce + 1) = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
        ctx = original;
        if(mine_single_fast(ctx, nonce, );

    } while ((hash_value < initial_hash_value + limit));

}

Nonce mine() {

}

void test_cuda() {
    int host_[N], hb[N];

    int *da, *db;
    if(hipMalloc((void **)&da, N*sizeof(int))){
        std::cout << "Failed to" << std::endl;
    }

    hipMalloc((void **)&db, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db);

    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);
}

